#include "hip/hip_runtime.h"
% % writefile matrixmul.cu

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 512
#define TOLERANCE 1e-6

        __global__ void
        matrixMul(float *A, float *B, float *C, int n)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n)
    {
        float sum = 0.0;
        for (int k = 0; k < n; k++)
        {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main()
{
    float *hA, *hB, *hC, *hGPU, *dA, *dB, *dC;
    size_t size = N * N * sizeof(float);
    hipEvent_t start, stop;
    float cpuTime, gpuTime;

    hA = (float *)malloc(size);
    hB = (float *)malloc(size);
    hC = (float *)malloc(size);
    hGPU = (float *)malloc(size);

    for (int i = 0; i < N * N; i++)
    {
        hA[i] = (float)rand() / RAND_MAX;
        hB[i] = (float)rand() / RAND_MAX;
    }

    clock_t cpustart = clock();
    float sum = 0.0;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < N; k++)
            {
                sum += hA[i * N + k] * hB[k * N + j];
            }
            hC[i * N + j] = sum;
            sum = 0.0;
        }
    }
    clock_t cpuend = clock();
    cpuTime = (float)(cpuend - cpustart) / CLOCKS_PER_SEC;
    printf("CPU time: %f ", cpuTime);

    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&dB, size);
    hipMalloc((void **)&dC, size);

    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    dim3 threadsperBlock(16, 16);
    dim3 blocksPerGrid((N + threadsperBlock.x - 1) / threadsperBlock.x, (N + threadsperBlock.y - 1) / threadsperBlock.y);

    matrixMul<<<blocksPerGrid, threadsperBlock>>>(dA, dB, dC, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpuTime, start, stop);
    gpuTime /= 1000;
    printf("GPU time: %f ", gpuTime);

    hipMemcpy(hGPU, dC, size, hipMemcpyDeviceToHost);

    int isValid = 1;
    for (int i = 0; i < N * N; i++)
    {
        if (fabs(hC[i] - hGPU[i]) > TOLERANCE * fabs(hC[i]))
        {
            isValid = 0;
            break;
        }
    }

    printf("Verification: %s\n", isValid ? "TRUE" : "FALSE");

    free(hA);
    free(hB);
    free(hC);
    free(hGPU);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}