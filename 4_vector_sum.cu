#include "hip/hip_runtime.h"
% % writefile vecAdd.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define TOLERANCE 1e-6

        __global__ void
        vectorAdd(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    float *hA, *hB, *hC, *hGPU, *dA, *dB, *dC;
    hipEvent_t start, stop;
    size_t size = N * sizeof(float);
    float gpuTime, cpuTime;

    hA = (float *)malloc(size);
    hB = (float *)malloc(size);
    hC = (float *)malloc(size);
    hGPU = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        hA[i] = rand() / (float)RAND_MAX;
        hB[i] = rand() / (float)RAND_MAX;
    }

    clock_t cpuStart = clock();

    for (int i = 0; i < N; i++)
    {
        hC[i] = hA[i] + hB[i];
    }
    clock_t cpuEnd = clock();
    cpuTime = (float)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;
    printf("CPU Time: %f\n", cpuTime);

    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&dB, size);
    hipMalloc((void **)&dC, size);

    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpuTime, start, stop);
    gpuTime /= 1000;

    hipMemcpy(hGPU, dC, size, hipMemcpyDeviceToHost);

    printf("GPU time: %f \n", gpuTime);

    bool isValid = true;

    for (int i = 0; i < N; i++)
    {
        if (fabs(hC[i] - hGPU[i]) > TOLERANCE)
        {
            isValid = false;
            break;
        }
    }

    printf("Verification: %s\n", isValid ? "TRUE" : "FALSE");

    free(hA);
    free(hB);
    free(hC);
    free(hGPU);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
